
#include <hip/hip_runtime.h>
typedef unsigned char uint8_t;
typedef unsigned long uint64_t;
__global__ void tsp(float *result, const float *arr, const uint8_t size, const uint8_t *perm, const uint64_t num_threads)
{
	/*Get global_id*/
        uint64_t block_id = blockIdx.x;
        uint64_t thread_id = threadIdx.x;
        uint64_t threads = blockDim.x;
        uint64_t global_id = thread_id + block_id * threads;
        if (global_id >= num_threads) return;
	/*Loop*/
	uint64_t perm_addon = global_id*size;
	float cost = 0;
	uint8_t lastp, currp;
	for (uint64_t i=0; i<size-1; i++)
	{	
		lastp = perm[perm_addon + i];
		currp = perm[perm_addon + i+1];
		cost += arr[lastp*size + currp];
	}
	{
		lastp = perm[perm_addon + size-1];
		currp = perm[perm_addon + 0];
		cost += arr[lastp*size + currp];
	}
	result[global_id] = cost;
}
