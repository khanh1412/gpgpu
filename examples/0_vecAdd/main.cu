
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include<ctime>

__global__ void add(float *s, float *a, float *b)
{
	uint64_t i = blockIdx.x;
	s[i] = a[i] + b[i];
}

const uint64_t COUNT = 10000000;
void ADD(float *s, float *a, float *b, uint64_t COUNT)
{

	float *ds; hipMalloc(&ds, COUNT*sizeof(float));
	float *da; hipMalloc(&da, COUNT*sizeof(float));
	float *db; hipMalloc(&db, COUNT*sizeof(float));

	hipStream_t queue; hipStreamCreate(&queue);
	{
	auto t1 = std::clock();

	hipMemcpyAsync(da, a, COUNT*sizeof(float), hipMemcpyHostToDevice, queue);
	hipMemcpyAsync(db, b, COUNT*sizeof(float), hipMemcpyHostToDevice, queue);
	dim3 global_dim(COUNT, 1, 1);
	dim3 local_dim(1, 1, 1);
	add<<<global_dim, local_dim>>>(ds, da, db);
	hipMemcpyAsync(s, ds, COUNT*sizeof(float), hipMemcpyDeviceToHost, queue);
	hipStreamSynchronize(queue);
	auto t2 = std::clock();
	std::cout<<"Total time: "<<(t2-t1)<<" ticks"<<std::endl;
	}
}
void print_array(float *a)
{
	for (uint64_t i=0; i<10; i++)
		std::cout<<a[i]<<"\t";
	std::cout<<"..."<<std::endl;
}
int main()
{
std::srand(std::time(nullptr));
	auto a = new float[COUNT];
	auto b = new float[COUNT];
	auto s = new float[COUNT];

	for (uint64_t i=0; i<COUNT; i++)
	{
		a[i] = static_cast<float>(std::rand()%10)/10;
		b[i] = static_cast<float>(std::rand()%10)/10;
		s[i] = 0;
	}
	ADD(s, a, b, COUNT);
	print_array(a);
	print_array(b);
	print_array(s);
	return 0;


}
