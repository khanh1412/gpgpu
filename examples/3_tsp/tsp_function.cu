
#include <hip/hip_runtime.h>
#include<cstdint>
#include<ctime>
uint64_t hm_permutations(uint8_t size);
__global__ void tsp(float *result, const float *arr, const uint8_t size, const uint8_t *perm, const uint64_t num_threads);
uint64_t cu_call(float *result, const float *arr, uint8_t size, const uint8_t *perm)
{
	std::clock_t t1, t2;
	/*SET UP*/
	hipStream_t stream;
	hipStreamCreate(&stream);
	const uint64_t num_threads = hm_permutations(size);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int max_block_dim = prop.maxThreadsPerBlock;
	//max_block_dim = 1;
	int hm_blocks = 1 + num_threads/max_block_dim;
	dim3 block_dim(max_block_dim);
	dim3 grid_dim(hm_blocks);
	/*BUFFER*/
	float *d_arr;    hipMalloc((void**)&d_arr,    size*size*sizeof(float));
	uint8_t *d_perm; hipMalloc((void**)&d_perm,   num_threads*size*sizeof(uint8_t));
	float *d_result; hipMalloc((void**)&d_result, num_threads*sizeof(float));
	/*KERNEL*/
	t1 = std::clock();
	hipMemcpyAsync(d_perm, perm,     num_threads*size*sizeof(uint8_t), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(d_arr,  arr,      size*size*sizeof(float), hipMemcpyHostToDevice,          stream);
	        tsp<<<grid_dim, block_dim, 0, stream>>>(d_result, d_arr, size, d_perm, num_threads);
	hipMemcpyAsync(result, d_result, num_threads*sizeof(float), hipMemcpyDeviceToHost,        stream);
	hipStreamSynchronize(stream);
	t2 = std::clock();
	/*CLEAN UP*/
	hipFree(d_arr);
	hipFree(d_perm);
	hipFree(d_result);
	hipStreamDestroy(stream); 
	return (1000000*(t2-t1))/CLOCKS_PER_SEC;
}
